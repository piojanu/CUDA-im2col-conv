
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>

// Feature maps dimensionality descriptions and assumptions:
//             : Height          : Width           : Channels  : Number                    :
// INPUT   / A | H               | W               | C         | ------------------------- |  
// KERNELS / F | P = K           | Q = K           | R = C     | D = number of kernels = 1 |
// OUTPUT  / B | L = H * (K - 1) | M = W * (K - 1) | N = D = 1 | ------------------------- |
// [!] K must be odd number.
// [!] Data layout for INPUT/OUTPUT: C x H x W.
// [!] Data layout for KERNELS: D x R(=C) x P(=K) x Q(=K)

// Turn on/off debug mode
// #define DEBUG
// #define FUNCTEST
#define PERFTEST

#ifdef DEBUG
    #define LOG(...) printf(__VA_ARGS__); fflush(stdout);
#else
    #define LOG(...) ;
#endif

const unsigned int H = 256, W = 128, C = 64, K = 3; 

// HOST FUNCTION
// Takes matrix A [double *matA] and transforms it
// into column representation [double *matAc]
void im2colOnHost(double *matA, double *matAc, int radiusF, int countLR, int L, int M, int K, int C)
{
    // For each spatial position in output...
    for (int m = 0; m < M; m++) {
        int w = m + radiusF;
        for (int l = 0; l < L; l++) {
            int h = l + radiusF;

            // Progress..
            LOG("\r[i] Calculation on CPU %3d%%...", ((m * L + l) * 100 / (M * L)));

            // For each kernel weight...
            for (int q = 0, oq = -1 * radiusF; oq <= radiusF; q++, oq++) {
                for (int p = 0, op = -1 * radiusF; op <= radiusF; p++, op++) {
                    for (int r = 0; r < C; r++) {
                        matAc[(l + L * m) + countLR * (r + C * (p + K * q))] = matA[r + C * ((h + op) + H * (w + oq))]; 
                        // LOG("matAc[%3d x %3d] <- matA[%3d x %3d x %3d]\n", (r + C * (p + K* q)), (l + L * m), (h + op), (w + oq), r);
                    }
                }
            }
        }
    }
    LOG("\n");
}
 
// DEVICE KERNEL
// Takes matrix A [double *matA] and transforms it
// into column representation [double *matAc] on GPU
__global__ 
void im2colOnDevice(unsigned int n, double *matAc, double *matA, int radiusF, int countLR, int L, int M, int K, int C)
{
    // Using grid-stride loop if too big problem size.
    // https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < n; 
         idx += blockDim.x * gridDim.x) 
    {
        int m = (idx / C) / L;
        int l = (idx / C) % L;
        int r = idx % C;
        
        // For each spatial position in output...
        if (m < M) {
            int w = m + radiusF;
            if (l < L) {
                int h = l + radiusF;
                // For each kernel weight...
                for (int q = 0, oq = -1 * radiusF; oq <= radiusF; q++, oq++) {
                    for (int p = 0, op = -1 * radiusF; op <= radiusF; p++, op++) {
                        if (r < C) {
                            matAc[(l + L * m) + countLR * (r + C * (p + K * q))] = matA[r + C * ((h + op) + H * (w + oq))]; 
                        }
                    }
                }
            }
        }
    }
}
 
// DEVICE KERNEL
// Takes matrix A [double *matA] and transforms it
// into column representation [double *matAc] on GPU
__global__ 
void col2imOnDevice(unsigned int n, double *matA, double *matAc, int radiusF, int countLR, int L, int M, int K, int C)
{
    // Using grid-stride loop if too big problem size.
    // https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < n; 
         idx += blockDim.x * gridDim.x) 
    {
        int m = (idx / C) / L;
        int l = (idx / C) % L;
        int r = idx % C;
    
        // For each spatial position in output...
        if (m < M) {
            int w = m + radiusF;
            if (l < L) {
                int h = l + radiusF;
                // For each kernel weight...
                for (int q = 0, oq = -1 * radiusF; oq <= radiusF; q++, oq++) {
                    for (int p = 0, op = -1 * radiusF; op <= radiusF; p++, op++) {
                        if (r < C) {
                            matA[r + C * ((h + op) + H * (w + oq))] = matAc[(l + L * m) + countLR * (r + C * (p + K * q))]; 
                        }
                    }
                }
            }
        }
    }
}

void program(unsigned int blockSize, unsigned int gridSize = 0)
{
    // CONSTS AND VARIABLES

    // Input/kernel/output counts and sizes
    const unsigned int countA = H*W*C;
    const size_t sizeA = countA*sizeof(double);
    LOG("[i] INPUT PARAMS: %u height, %u width, %u channels, %u elems, %u bytes\n", H, W, C, countA, sizeA);

    const unsigned int radiusF = (K - 1) / 2;
    const unsigned int countF = K*K*C;
    LOG("[i] FILTER PARAMS: %u radius, %u elems, %u bytes\n", radiusF, countF, sizeF);
    LOG("[i] FILTERS PARAMS: %u elems, %u bytes\n", countFs, sizeFs);
    
    const unsigned int L = H - (K - 1);
    const unsigned int M = W - (K - 1);
    LOG("[i] OUTPUT PARAMS: %u height, %u width, %u channels\n", L, M, D);
    
    const unsigned int countLR = L * M;
    const unsigned int countAc = countF * countLR;
    const size_t sizeAc = countAc*sizeof(double);
    LOG("[i] INPUT IN COL PARAMS: %u elems, %u bytes\n", countAc, sizeAc);

    
    // PREPARE DATA

    // Generate input data
    double *matA = (double *)malloc(sizeA);
    for (int i = 0; i < countA; i++) {
        matA[i] = i;
    }
    LOG("  [!] FINISHED GENERATING INPUT\n");

#ifdef FUNCTEST
    // Calculate im2col result
    double *matAc = (double *)malloc(sizeAc);
    im2colOnHost(matA, matAc, radiusF, countLR, L, M, K, C);
    LOG("  [!] FINISHED CALCULATING im2col RESULT ON CPU\n");
#endif


    // Alloc memory and copy data to device
    double *devA, *devAc, *retAc;
    
    hipMalloc((void**)&devA, sizeA); 
    hipMalloc((void**)&devAc, sizeAc); 
    retAc = (double *)malloc(sizeAc);

    hipMemcpy(devA, matA, sizeA, hipMemcpyHostToDevice); 

    // Compute default grid size if it wasn't passed
    const unsigned int KERNELS_NUM = L * M * C;
    if (gridSize == 0)
        gridSize = (KERNELS_NUM + blockSize - 1) / blockSize;
    
    // Run im2col computation on device and copy results
    im2colOnDevice<<<gridSize, blockSize>>>(KERNELS_NUM, devAc, devA, radiusF, countLR, L, M, K, C);
    LOG("  [!] FINISHED CALCULATING im2col ON DEVICE\n");
    
    hipMemcpy(retAc, devAc, sizeAc, hipMemcpyDeviceToHost);

#ifdef FUNCTEST
    // Compare results
    int success = 1;
    for (int i = 0; i < countAc; i++) {
        if (retAc[i] != matAc[i]) {
            success = 0;
            printf("TEST FAILED: im2col device kernel...\n");
            break;
        }
    }

    if (success) {
        printf("TEST PASSED: im2col device kernel!\n");
    }
#endif

    // Allocate memory for return value
    double *retA;
    retA = (double *)malloc(sizeA);
    hipMemset(devA, 0, sizeA); 
    
    // Run col2im computation on device and copy results
    col2imOnDevice<<<gridSize, blockSize>>>(KERNELS_NUM, devA, devAc, radiusF, countLR, L, M, K, C);
    LOG("  [!] FINISHED CALCULATING col2im ON DEVICE\n");
    
    hipMemcpy(retA, devA, sizeA, hipMemcpyDeviceToHost);

#ifdef FUNCTEST
    // Compare results
    success = 1;
    for (int i = 0; i < countA; i++) {
        if (retA[i] != matA[i]) {
            success = 0;
            printf("TEST FAILED: col2im device kernel...\n");
            break;
        }
    }

    if (success) {
        printf("TEST PASSED: col2im device kernel!\n");
    }
#endif

    // CLEAN UP
    hipFree(devA);
    hipFree(devAc);
    
    free(matA);
#ifdef FUNCTEST
    free(matAc);
#endif
    free(retA);
    free(retAc);
}

int main()
{
    // Enforce default grid size
    unsigned int gridSize = 0;
    
    // First warm-up run
    program(256);

#ifdef PERFTEST
    // Set grid size
    gridSize = 1;
    
    // Open file for perf logs
    std::fstream fperflog("perflog.csv", std::ios::out);
    if (fperflog.good())
    {
        // Measure effect of different block sizes
        for (unsigned int blockSize = 2; blockSize <= 2048; blockSize *= 2) {
#endif

            struct timeval t1, t2;
            double elapsedTime, totalTime = 0;
            const int totalRuns = 10;
            
            for (int i = 0; i < totalRuns; i++) {
                // Start timer
                gettimeofday(&t1, NULL);
                
                // WORK HARD!
                program(blockSize, gridSize);
                
                // Stop timer
                gettimeofday(&t2, NULL);
                
                // Compute the elapsed time in millisec
                elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
                elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
                
                totalTime += elapsedTime;
            }
            LOG("  [!] Whole program took %.3fms averaged over %d runs\n", totalTime / totalRuns, totalRuns);
            
#ifdef PERFTEST
            fperflog << blockSize << "," << gridSize << "," << elapsedTime << std::endl;
        }
        
        // Close file
        fperflog.close();
    }
#endif

    return EXIT_SUCCESS;
}